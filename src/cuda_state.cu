#include "cuda_state.hpp"

#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vulkan/vulkan.h>

#include "utils.hpp"

using namespace std;

namespace v4r {

hipExternalMemory_t importBuffer(int buf_fd, uint64_t num_bytes)
{
    hipExternalMemoryHandleDesc cuda_ext_info {};
    cuda_ext_info.type = hipExternalMemoryHandleTypeOpaqueFd;
    cuda_ext_info.handle.fd = buf_fd;
    cuda_ext_info.size = num_bytes;
    cuda_ext_info.flags = hipExternalMemoryDedicated;

    hipExternalMemory_t ext_mem;
    hipError_t res = hipImportExternalMemory(&ext_mem, &cuda_ext_info);

    if (res != hipSuccess) {
        cerr << "CUDA failed to import vulkan buffer" << endl;
        fatalExit();
    }

    return ext_mem;
}

void *mapExternal(hipExternalMemory_t ext_mem, uint64_t num_bytes)
{
    void *dev_ptr;
    hipExternalMemoryBufferDesc ext_info;
    ext_info.offset = 0;
    ext_info.size = num_bytes;
    ext_info.flags = 0;

    hipError_t res = hipExternalMemoryGetMappedBuffer(&dev_ptr, ext_mem, &ext_info);
    if (res != hipSuccess) {
        cerr << "CUDA failed to map vulkan buffer" << endl;
        fatalExit();
    }

    return dev_ptr;
}

CudaState::CudaState(int buf_fd, uint64_t num_bytes)
    : cuda_ext_mem_(importBuffer(buf_fd, num_bytes)),
      dev_ptr_(mapExternal((hipExternalMemory_t)cuda_ext_mem_, num_bytes))
{}

}
